#include "hip/hip_runtime.h"
//
// Created by alfonso on 12/10/21.
//

#include "../distlocRutaCUDA.h"

/**
 *
 * @param d_pseg
 * @param d_preln
 * @param d_pidnodo
 * @param tam_nodo
 * @param tam_seg
 */
__global__ void
kernel_generaRed(PSegmento d_pseg, PDRelNodo d_preln,const long *d_pidnodo, unsigned int tam_seg, unsigned int tam_nodo) {

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= tam_nodo)return;

    int n = 0;

    for (unsigned int iseg = 0; iseg < tam_seg; iseg++) {

        if ((d_pseg + iseg)->id_nodo1 == *(d_pidnodo + idx)) {
            (d_preln + idx)->aid_nodos[n] = (d_pseg + iseg)->id_nodo0;
            (d_preln + idx)->adist[n] = (d_pseg + iseg)->dist;
            n++;
        } else if ((d_pseg + iseg)->id_nodo0 == *(d_pidnodo + idx)) {
            (d_preln + idx)->aid_nodos[n] = (d_pseg + iseg)->id_nodo1;
            (d_preln + idx)->adist[n] = (d_pseg + iseg)->dist;
            n++;
        }

    }

    (d_preln + idx)->nsn = n;
}


/**
 *
 * @param h_pseg
 * @param pnodo
 * @param tam_seg
 * @param tam_nodo
 */
void generaRedNodos(PSegmento h_pseg, PNodo pnodo, unsigned int tam_seg, unsigned int tam_nodo) {

    std::cout << "Relaciona Nodos para formar Red ..." << std::endl;

    int canti_hilos = 1000;
    int canti_bloques = (int) ceil((double) tam_nodo / canti_hilos);


    std::cout << "Hilos: " << canti_hilos << " Bloques: " << canti_bloques << std::endl;

    long *d_pidnodo;
    PDRelNodo d_preln;
    PSegmento d_pseg;

    auto h_pidnodo = (long *) malloc(sizeof(long) * tam_nodo);
    auto h_preln = (PDRelNodo) malloc(sizeof(struct DRelNodo) * tam_nodo);

    for (size_t i = 0; i < tam_nodo; i++) {
        *(h_pidnodo + i) = (pnodo + i)->id_nodo;
    }

    hipMalloc((void **) &(d_pseg), tam_seg * sizeof(struct Segmento));
    hipMalloc((void **) &(d_preln), tam_nodo * sizeof(struct DRelNodo));
    hipMalloc((void **) &(d_pidnodo), tam_nodo * sizeof(long));

    hipMemcpy(d_pidnodo, h_pidnodo, tam_nodo * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(d_pseg, h_pseg, tam_seg * sizeof(struct Segmento), hipMemcpyHostToDevice);

    kernel_generaRed<<<canti_bloques, canti_hilos>>>(d_pseg, d_preln,(const long*) d_pidnodo, tam_seg, tam_nodo);

    hipMemcpy(h_preln, d_preln, tam_nodo * sizeof(struct DRelNodo), hipMemcpyDeviceToHost);

    for (auto i = 0; i < tam_nodo; i++) {
        PDRelNodo prel = h_preln + i;
        if (prel->nsn > 0) {
            PNodo pn = pnodo + i;
            pn->nsn = prel->nsn;
            for (int j = 0; j < pn->nsn; j++) {
                pn->adist[j] = prel->adist[j];
                pn->aid_nodos[j] = prel->aid_nodos[j];
            }
        }
    }

    hipFree(d_pidnodo);
    hipFree(d_preln);
    hipFree(d_pseg);

    free(h_preln);
    free(h_pidnodo);
}