#include "hip/hip_runtime.h"
//
// Created by alfonso on 14/09/21.
//
#include "../distlocRutaCUDA.h"
#include "../distlocRuta.h"


void relacionaLoc2Nodo(PLocalidad ploc, PNodo pnodo, unsigned int tam_loc, unsigned int tam_nodo);



/**
 *
 * @param d_pnodo
 * @param d_ploc
 * @param d_presu
 * @param tam_loc
 * @param tam_n0
 * @param tam_n1
 */
__global__ void
calculadistL2Nv2(PDNodo d_pnodo, PDLocalidad d_ploc, PDResu d_presu, unsigned int tam_loc, unsigned int tam_n0,
                 unsigned int tam_n1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double dmin = 8;
    long jmin = -1;
    __shared__ double daux;

    if (idx >= tam_loc)return;
    PDLocalidad p = d_ploc + idx;

    if (tam_n0 > 0) {
        dmin = (d_presu + idx)->dist;
        jmin = (d_presu + idx)->index;
    }

    for (unsigned int j = tam_n0; j < tam_n1; j++) {
        PDNodo pn = d_pnodo + j;
        daux = pn->x * p->x + pn->y * p->y + pn->z * p->z;
        if (daux > 1.00)daux = 1.00;
        if (daux < -1.00)daux = -1.00;
        daux = acos(daux);
        if (dmin > daux) {
            dmin = daux;
            jmin = j;
        }
    }

    (d_presu + idx)->dist = dmin;
    (d_presu + idx)->index = jmin;

}




/**
 *
 * @param ploc
 * @param pnodo
 * @param tam_loc
 * @param tam_nodo
 */
void relacionaLoc2Nodo(PLocalidad ploc, PNodo pnodo, unsigned int tam_loc, unsigned int tam_nodo) {

    std::cout << "Relaciona Localidades con Nodos ..." << std::endl;


    int canti_hilos = 1000;
    int canti_bloques = (int) ceil((double) tam_loc / canti_hilos) + 1;


    unsigned int delta = 200000;
    unsigned int tam_n1;

    std::cout << "Hilos: " << canti_hilos << " Bloques: " << canti_bloques << std::endl;

    PDNodo d_pnodo = nullptr;
    PDLocalidad d_ploc = nullptr;
    PDResu d_presu = nullptr;

    auto h_pnodo = (PDNodo) malloc(sizeof(struct DNodo) * tam_nodo);
    auto h_ploc = (PDLocalidad) malloc(sizeof(struct DLocalidad) * tam_loc);
    auto h_presu = (PDResu) malloc(sizeof(struct DResu) * tam_loc);

    for (auto i = 0; i < tam_loc; i++) {
        (h_ploc + i)->x = (ploc + i)->x;
        (h_ploc + i)->y = (ploc + i)->y;
        (h_ploc + i)->z = (ploc + i)->z;
    }

    for (auto i = 0; i < tam_nodo; i++) {
        (h_pnodo + i)->x = (pnodo + i)->x;
        (h_pnodo + i)->y = (pnodo + i)->y;
        (h_pnodo + i)->z = (pnodo + i)->z;
    }

    hipMalloc((void **) &(d_ploc), tam_loc * sizeof(struct DLocalidad));
    hipMalloc((void **) &(d_pnodo), tam_nodo * sizeof(struct DNodo));
    hipMalloc((void **) &(d_presu), tam_loc * sizeof(struct DResu));

    hipMemcpy(d_ploc, h_ploc, tam_loc * sizeof(struct DLocalidad), hipMemcpyHostToDevice);
    hipMemcpy(d_pnodo, h_pnodo, tam_nodo * sizeof(struct DNodo), hipMemcpyHostToDevice);


    int ii = 1;
    for (unsigned int tam_n0 = 0; tam_n0 < tam_nodo; tam_n0 += delta) {
        tam_n1 = (tam_n0 + delta > tam_nodo) ? tam_nodo : tam_n0 + delta;
        std::cout << "ciclo: " << ii << " " << tam_n0 << " " << tam_n1 << std::endl;
        ii++;
        calculadistL2Nv2<<<canti_bloques, canti_hilos>>>(d_pnodo, d_ploc, d_presu, tam_loc, tam_n0, tam_n1);
        hipDeviceSynchronize();
    }


    hipMemcpy(h_presu, d_presu, tam_loc * sizeof(struct DResu), hipMemcpyDeviceToHost);

    for (auto i = 0; i < tam_loc; i++) {
        (ploc + i)->dist_nodo = (h_presu + i)->dist;
        (ploc + i)->index_nodo = (h_presu + i)->index;
        (ploc + i)->id_nodo = (pnodo + (ploc + i)->index_nodo)->id_nodo;
    }

    hipFree(d_presu);
    hipFree(d_pnodo);
    hipFree(d_ploc);

    free(h_presu);
    free(h_ploc);
    free(h_pnodo);
}