#include "hip/hip_runtime.h"
//
// Created by alfonso on 25/09/21.
//

#include "../distlocRutaCUDA.h"

/**
 * Función que marca subnodos con distancia
 * @param d_pnodored
 * @param d_psubnodo
 * @param d_presrec
 * @param idx
 * @param dist
 * @param irec
 */
__device__ void
recorrido(PNodoRed d_pnodored, PSubNodo d_psubnodo, PResRec d_presrec, unsigned int idx, double dist, long irec) {

    if ((d_psubnodo + idx)->nsn == 0)return;

    for (int i = 0; i < (d_psubnodo + idx)->nsn; i++) {

        unsigned int index = (d_psubnodo + idx)->aindex_nodos[i];
        if (index != idx) {
            double nueva_dist = (d_pnodored + idx)->adist[i] + dist;

            if ((d_presrec + index)->id_rec < 0 ||
                ((d_presrec + index)->id_rec > 0 && (d_presrec + index)->dist_rec > nueva_dist)) {

                (d_presrec + index)->delta += nueva_dist - (d_presrec + index)->dist_rec;
                (d_presrec + index)->dist_rec = nueva_dist;
                (d_presrec + index)->id_rec = irec;

            }
        }
    }
}


/**
 * Kernel principal para marcar y propagar las distancias desde los nodos que poseen un recurso
 * @param d_pnodored
 * @param d_psubnodo
 * @param d_presrec
 * @param tam_nodo
 */
__global__ void kernel_principalDN(PNodoRed d_pnodored, PSubNodo d_psubnodo, PResRec d_presrec, unsigned int tam_nodo) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= tam_nodo)return;

    if ((d_presrec + idx)->id_rec > 0) {
        recorrido(d_pnodored, d_psubnodo, d_presrec, idx, (d_presrec + idx)->dist_rec, (d_presrec + idx)->id_rec);
    }
}
