#include "hip/hip_runtime.h"
//
// Created by alfonso on 25/09/21.
//

#include "../distlocRutaCUDA.h"

void generaRutas(PNodo pnodo, unsigned int tam_nodo);

extern __global__ void
kernel_principalDN(PNodoRed d_pnodored, PSubNodo d_psubnodo, PResRec d_presrec, unsigned int tam_nodo);

extern double sumaVariacion(PResRec d_presrec, unsigned int tam_nodo);




/**
 * Kernel que recupera los indices para los identificadores de los subnodos
 * @param d_pnodored
 * @param d_psubnodo
 * @param d_pidnodo
 * @param tam_nodo
 */
__global__ void
kernel_indexaNodos(PNodoRed d_pnodored, PSubNodo d_psubnodo, const long *d_pidnodo, unsigned int tam_nodo,
                   unsigned int tam_n0, unsigned int tam_n1) {

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= tam_nodo)return;

    PNodoRed pnr = d_pnodored + idx;

    for (int i = 0; i < pnr->nsn; i++) {
        for (unsigned int j = tam_n0; j < tam_n1; j++) {
            if (*(d_pidnodo + j) == pnr->aid_nodos[i]) {
                (d_psubnodo + idx)->aindex_nodos[i] = j;
                break;
            }
        }
    }
    (d_psubnodo + idx)->nsn = pnr->nsn;
}


/**
 * Kernel que prepara el arreglo de resultados distancias ids, dist
 * @param d_pnodored
 * @param d_presrec
 * @param tam_nodo
 */
__global__ void kernel_preparaDist(PNodoRed d_pnodored, PResRec d_presrec, unsigned int tam_nodo) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= tam_nodo)return;

    (d_presrec + idx)->id_rec = (d_pnodored + idx)->id_rec;
    (d_presrec + idx)->dist_rec = (d_pnodored + idx)->dist_rec;
    (d_presrec + idx)->delta = 0.0;
}


/**
 *
 * @param pnodo
 * @param tam_nodo
 */
void generaRutas(PNodo pnodo, unsigned int tam_nodo) {

    int canti_hilos = 1000;
    int canti_bloques = (int) ceil((double) tam_nodo / canti_hilos);

    std::cout << "Bloques: " << canti_bloques << " Hilos: " << canti_hilos << std::endl;

    auto h_pnodored = (PNodoRed) malloc(sizeof(struct NodoRed) * tam_nodo);
    auto h_pidnodo = (long *) malloc(sizeof(long) * tam_nodo);
    auto h_psubnodo = (PSubNodo) malloc(sizeof(struct SubNodo) * tam_nodo);
    auto h_presrec = (PResRec) malloc(sizeof(struct ResRec) * tam_nodo);


    PNodoRed d_pnodored;
    long *d_pidnodo;
    PSubNodo d_psubnodo;
    PResRec d_presrec;


    for (unsigned int i = 0; i < tam_nodo; i++) {
        PNodo p = pnodo + i;

        if (p->id_rec > 0) {
            (h_pnodored + i)->id_rec = p->id_rec;
            (h_pnodored + i)->dist_rec = p->dist_rec;
        } else {
            (h_pnodored + i)->id_rec = -1;
            (h_pnodored + i)->dist_rec = MAX_DIST;
        }

        (h_pnodored + i)->nsn = 0;
        if (p->nsn > 0) {
            for (int j = 0; j < p->nsn; j++) {
                (h_pnodored + i)->aid_nodos[j] = p->aid_nodos[j];
                (h_pnodored + i)->adist[j] = p->adist[j];
            }
            (h_pnodored + i)->nsn = p->nsn;
        }

        *(h_pidnodo + i) = p->id_nodo;

    }

    hipMalloc((void **) &(d_pnodored), tam_nodo * sizeof(struct NodoRed));
    hipMalloc((void **) &(d_pidnodo), tam_nodo * sizeof(long));
    hipMalloc((void **) &(d_psubnodo), tam_nodo * sizeof(struct SubNodo));
    hipMalloc((void **) &(d_presrec), tam_nodo * sizeof(struct ResRec));


    hipMemcpy(d_pnodored, h_pnodored, tam_nodo * sizeof(struct NodoRed), hipMemcpyHostToDevice);
    hipMemcpy(d_pidnodo, h_pidnodo, tam_nodo * sizeof(long), hipMemcpyHostToDevice);

    std::cout << "Vuelve indices a id_nodos ..." << std::endl;


    int ii = 1;
    unsigned int delta = 200000;
    unsigned int tam_n1;

    for (unsigned int tam_n0 = 0; tam_n0 < tam_nodo; tam_n0 += delta) {
        tam_n1 = (tam_n0 + delta > tam_nodo) ? tam_nodo : tam_n0 + delta;
        std::cout << "ciclo: " << ii << " " << tam_n0 << " " << tam_n1 << std::endl;
        ii++;
        kernel_indexaNodos<<<canti_bloques, canti_hilos>>>(d_pnodored, d_psubnodo, d_pidnodo, tam_nodo, tam_n0, tam_n1);
        hipDeviceSynchronize();
    }


    std::cout << "Prepara el destino para registrar las distancias ..." << std::endl;

    kernel_preparaDist<<<canti_bloques, canti_hilos>>>(d_pnodored, d_presrec, tam_nodo);
    hipDeviceSynchronize();


    std::cout << "Cálcula distancias a nodos ..." << std::endl;

    double variacion1, variacion0;
    int numiters = 600;
    do {
        variacion0 = variacion1;
        kernel_principalDN<<<canti_bloques, canti_hilos>>>(d_pnodored, d_psubnodo, d_presrec, tam_nodo);
        variacion1 = sumaVariacion(d_presrec, tam_nodo);
        std::cout << "Ciclo cálculo:" << numiters << " " << variacion0 - variacion1 << std::endl;
        numiters--;
    } while (variacion0 != variacion1 && numiters > 0);


    hipMemcpy(h_presrec, d_presrec, tam_nodo * sizeof(struct ResRec), hipMemcpyDeviceToHost);
    hipMemcpy(h_psubnodo, d_psubnodo, tam_nodo * sizeof(struct SubNodo), hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < tam_nodo; i++) {
        (pnodo + i)->id_rec = (h_presrec + i)->id_rec;
        (pnodo + i)->dist_rec = (h_presrec + i)->dist_rec;

        for (int j = 0; j < (h_psubnodo + i)->nsn; j++) {
            (pnodo + i)->aindex_nodos[j] = (h_psubnodo + i)->aindex_nodos[j];
        }
    }


    hipFree(d_pnodored);
    hipFree(d_psubnodo);
    hipFree(d_pidnodo);
    hipFree(d_presrec);

    free(h_pnodored);
    free(h_pidnodo);
    free(h_psubnodo);
    free(h_presrec);
}


