#include "hip/hip_runtime.h"
//
// Created by alfonso on 16/09/21.
//
#include "../distlocRutaCUDA.h"

const static double RT = 6371000.00;

void relacionaRec2Nodo(PRecurso prec, PNodo pnodo, unsigned int tam_rec, unsigned int tam_nodo);

/**
 *
 * @param d_pnodo
 * @param d_prec
 * @param d_presu
 * @param tam_loc
 * @param tam_n0
 * @param tam_n1
 */
__global__ void
calculadistR2Nv2(PDNodo d_pnodo, PDRecurso d_prec, PDResu d_presu, unsigned int tam_loc, unsigned int tam_n0,
                 unsigned int tam_n1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double dmin = 8;
    long jmin = -1;

    if (idx >= tam_loc)return;
    PDRecurso p = d_prec + idx;

    if (tam_n0 > 0) {
        dmin = (d_presu + idx)->dist;
        jmin = (d_presu + idx)->index;
    }

    for (unsigned int j = tam_n0; j < tam_n1; j++) {
        PDNodo pn = d_pnodo + j;
        double daux = pn->x * p->x + pn->y * p->y + pn->z * p->z;
        if (daux > 1.00)daux = 1.00;
        if (daux < -1.00)daux = -1.00;
        daux = acos(daux);
        if (dmin > daux) {
            dmin = daux;
            jmin = j;
        }
    }

    (d_presu + idx)->dist = RT * dmin;
    (d_presu + idx)->index = jmin;
}

/**
 *
 * @param prec
 * @param pnodo
 * @param tam_rec
 * @param tam_nodo
 */
void relacionaRec2Nodo(PRecurso prec, PNodo pnodo, unsigned int tam_rec, unsigned int tam_nodo) {

    std::cout << "Relaciona Recursos con Nodos ..." << std::endl;

    int canti_hilos = 840;
    int canti_bloques = (int) ceil((double) tam_rec / canti_hilos) + 1;

    unsigned int delta = 100000;
    unsigned int tam_n1;

    std::cout << "Hilos: " << canti_hilos << " Bloques: " << canti_bloques << std::endl;

    PDNodo d_pnodo = nullptr;
    PDRecurso d_prec = nullptr;
    PDResu d_presu = nullptr;


    auto h_pnodo = (PDNodo) malloc(sizeof(struct DNodo) * tam_nodo);
    auto h_prec = (PDRecurso) malloc(sizeof(struct DRecurso) * tam_rec);
    auto h_presu = (PDResu) malloc(sizeof(struct DResu) * tam_rec);

    for (auto i = 0; i < tam_rec; i++) {
        (h_prec + i)->x = (prec + i)->x;
        (h_prec + i)->y = (prec + i)->y;
        (h_prec + i)->z = (prec + i)->z;
    }

    for (auto i = 0; i < tam_nodo; i++) {
        (h_pnodo + i)->x = (pnodo + i)->x;
        (h_pnodo + i)->y = (pnodo + i)->y;
        (h_pnodo + i)->z = (pnodo + i)->z;
    }

    hipMalloc((void **) &(d_prec), tam_rec * sizeof(struct DRecurso));
    hipMalloc((void **) &(d_pnodo), tam_nodo * sizeof(struct DNodo));
    hipMalloc((void **) &(d_presu), tam_rec * sizeof(struct DResu));

    hipMemcpy(d_prec, h_prec, tam_rec * sizeof(struct DRecurso), hipMemcpyHostToDevice);
    hipMemcpy(d_pnodo, h_pnodo, tam_nodo * sizeof(struct DNodo), hipMemcpyHostToDevice);

    int ii = 1;
    for (unsigned int tam_n0 = 0; tam_n0 < tam_nodo; tam_n0 += delta) {
        tam_n1 = (tam_n0 + delta > tam_nodo) ? tam_nodo : tam_n0 + delta;
        std::cout << "ciclo: " << ii << " " << tam_n0 << " " << tam_n1 << std::endl;
        ii++;
        calculadistR2Nv2<<<canti_bloques, canti_hilos>>>(d_pnodo, d_prec, d_presu, tam_rec, tam_n0, tam_n1);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_presu, d_presu, tam_rec * sizeof(struct DResu), hipMemcpyDeviceToHost);

    for (auto i = 0; i < tam_rec; i++) {
        (prec + i)->dist_nodo = (h_presu + i)->dist;
        (prec + i)->index_nodo = (h_presu + i)->index;
        (prec + i)->id_nodo = (pnodo + (prec + i)->index_nodo)->id_nodo;
        (pnodo + (prec + i)->index_nodo)->id_rec = (prec + i)->id_rec;
        (pnodo + (prec + i)->index_nodo)->dist_rec = (prec + i)->dist_nodo;
    }

    hipFree(d_presu);
    hipFree(d_pnodo);
    hipFree(d_prec);

    free(h_presu);
    free(h_prec);
    free(h_pnodo);
}
