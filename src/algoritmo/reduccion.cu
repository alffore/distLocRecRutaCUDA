#include "hip/hip_runtime.h"
//
// Created by alfonso on 26/09/21.
//
#include "../distlocRutaCUDA.h"

static const unsigned int blockSize = 1024;
static const unsigned int gridSize = 24;

/**
 *
 * @param d_presrec
 * @param tam_nodo
 * @param d_aOut
 */
__global__ void kernel_sumaVariaciones(PResRec d_presrec, unsigned int tam_nodo, double *d_aOut) {

    unsigned int tid = threadIdx.x;
    unsigned int gtid = tid + blockIdx.x * blockSize;
    const unsigned int gsize = blockSize * gridDim.x;

    double sum = 0;

    for (unsigned int i = gtid; i < tam_nodo; i += gsize) {
        sum += (d_presrec + i)->delta;
    }

    __shared__ double shArr[blockSize];
    shArr[tid] = sum;
    __syncthreads();

    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (tid < size)
            shArr[tid] += shArr[tid + size];

        __syncthreads();
    }

    if (tid == 0)
        d_aOut[blockIdx.x] = shArr[0];
}

/**
 *
 * @param d_aIn
 * @param tam_nodo
 * @param d_aOut
 */
__global__ void kernel_sumaVariaciones2(double *d_aIn, unsigned int tam_nodo, double *d_aOut) {

    unsigned int tid = threadIdx.x;
    unsigned int gtid = tid + blockIdx.x * blockSize;
    const unsigned int gsize = blockSize * gridDim.x;

    double sum = 0;

    for (unsigned int i = gtid; i < tam_nodo; i += gsize) {
        sum += d_aIn[i];
    }

    __shared__ double shArr[blockSize];
    shArr[tid] = sum;
    __syncthreads();

    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (tid < size)
            shArr[tid] += shArr[tid + size];

        __syncthreads();
    }

    if (tid == 0)
        d_aOut[blockIdx.x] = shArr[0];
}


/**
 *
 * @param d_presrec
 * @param tam_nodo
 * @return
 */
double sumaVariacion(PResRec d_presrec,unsigned int tam_nodo){

    double out;
    double *d_aOut;

    hipMalloc((void **)&d_aOut,sizeof(double)*gridSize);

    kernel_sumaVariaciones<<<gridSize,blockSize>>>(d_presrec,tam_nodo,d_aOut);

    kernel_sumaVariaciones2<<<1,blockSize>>>(d_aOut,gridSize,d_aOut);

    hipDeviceSynchronize();

    hipMemcpy(&out, d_aOut, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_aOut);

    return out;
}
