#include "hip/hip_runtime.h"
//
// Created by alfonso on 14/09/21.
//
#include "../distlocRuta.h"

int cargaArchivoNodo(std::string snomarchivo, PNodo pnodo);

void parseaLineaNodo(char *slinea, PNodo pnodo);

void convierteCoordNodo(PNodo,double,double);

/**
 *
 * @param snomarchivo
 * @param pnodo
 * @return
 */
int cargaArchivoNodo(std::string snomarchivo, PNodo pnodo) {

    printf("Carga nodos ...\n");

    FILE *fp = fopen(snomarchivo.c_str(), "r");
    unsigned int pos = 0;

    if (fp != NULL) {
        char buffer[MAX_LONG];

        while (fgets(buffer, MAX_LONG, fp)) {
            parseaLineaNodo(buffer, pnodo + pos);
            pos++;
        }
    }

    fclose(fp);
    return 0;
}

/**
 *
 * @param slinea
 * @param pnodo
 */
void parseaLineaNodo(char *slinea, PNodo pnodo) {
    char *result = NULL;
    double lon,lat;

    result = strtok(slinea, SEP);

    lon = atof(result);
    result = strtok(NULL, SEP);

    lat = atof(result);
    result = strtok(NULL, SEP);

    pnodo->id_nodo = atoi(result);


    pnodo->id_rec = -1;

    for (int i = 0; i < SND; i++) {
        pnodo->adist[i] = -1.0;
        pnodo->aid_nodos[i] = -1;
    }

    pnodo->nsn = 0;



    convierteCoordNodo(pnodo,lon,lat);
}

/**
 *
 * @param p
 */
void convierteCoordNodo(PNodo p,double lon,double lat) {

    double r;

    lat = lat * M_PI / 180.0;
    lon = lon * M_PI / 180.0;

    p->x = sin(lon) * cos(lat);
    p->y = cos(lon) * cos(lat);
    p->z = sin(lat);

    r = pow(p->x, 2.0) + pow(p->y, 2.0) + pow(p->z, 2.0);
    r = sqrt(r);

    p->x /= r;
    p->y /= r;
    p->z /= r;
}
