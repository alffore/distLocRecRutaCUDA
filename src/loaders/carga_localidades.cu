#include "hip/hip_runtime.h"
//
// Created by alfonso on 14/09/21.
//

#include "../distlocRuta.h"

int cargaArchivoLoc(std::string snomarchivo, PLocalidad ploc);

void parseaLineaLoc(char *slinea, PLocalidad ploc);

void convierteCoordLoc(PLocalidad p, double lon, double lat);


/**
 * Función que carga el archivo de Localidades
 */
int cargaArchivoLoc(std::string snomarchivo, PLocalidad ploc) {

    printf("Carga localidades ...\n");

    FILE *fp = fopen(snomarchivo.c_str(), "r");
    unsigned int pos = 0;

    if (fp != NULL) {
        char buffer[MAX_LONG];

        while (fgets(buffer, MAX_LONG, fp)) {
            parseaLineaLoc(buffer, ploc + pos);
            pos++;
        }
    }

    fclose(fp);
    return 0;
}

/**
 *
 */
void parseaLineaLoc(char *slinea, PLocalidad ploc) {
    char *result = NULL;
    double lon,lat;

    result = strtok(slinea, SEP);

    lon = atof(result);
    result = strtok(NULL, SEP);

    lat = atof(result);
    result = strtok(NULL, SEP);

    result = strtok(NULL, SEP);

    ploc->id_loc = atoi(result);

    convierteCoordLoc(ploc,lon,lat);


    ploc->id_nodo = -1;
    ploc->dist_nodo = MAX_DIST;
    ploc->index_nodo=-1;
}


/**
 *
 */
void convierteCoordLoc(PLocalidad p, double lon, double lat) {

    double r;

    lat = lat * M_PI / 180.0;
    lon = lon * M_PI / 180.0;

    p->x = sin(lon) * cos(lat);
    p->y = cos(lon) * cos(lat);
    p->z = sin(lat);

    r = pow(p->x, 2.0) + pow(p->y, 2.0) + pow(p->z, 2.0);
    r = sqrt(r);

    p->x /= r;
    p->y /= r;
    p->z /= r;
}

