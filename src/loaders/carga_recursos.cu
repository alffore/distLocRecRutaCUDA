#include "hip/hip_runtime.h"
//
// Created by alfonso on 14/09/21.
//

#include "../distlocRuta.h"

int cargaArchivoRec(std::string snomarchivo, PRecurso prec);

void parseaLineaRec(char *slinea, PRecurso prec);

void convierteCoordRec(PRecurso p, double lon, double lat);

/**
 *
 * @param snomarchivo
 * @param prec
 * @return
 */
int cargaArchivoRec(std::string snomarchivo, PRecurso prec) {

    printf("Carga Recursos ...\n");

    FILE *fp = fopen(snomarchivo.c_str(), "r");
    unsigned int pos = 0;

    if (fp != NULL) {
        char buffer[MAX_LONG];

        while (fgets(buffer, MAX_LONG, fp)) {
            parseaLineaRec(buffer, prec + pos);
            pos++;
        }
    }

    fclose(fp);
    return 0;
}

/**
 *
 * @param slinea
 * @param prec
 */
void parseaLineaRec(char *slinea, PRecurso prec){
    char *result = NULL;
    double lon,lat;
    result = strtok(slinea, SEP);

    lon = atof(result);
    result = strtok(NULL, SEP);

    lat = atof(result);
    result = strtok(NULL, SEP);

    prec->id_rec = atoi(result);

    convierteCoordRec(prec,lon,lat);

    prec->dist_nodo = MAX_DIST;
    prec->id_nodo = -1;
    prec->index_nodo=-1;
    
}

/**
 *
 * @param p
 * @param lon
 * @param lat
 */
void convierteCoordRec(PRecurso p, double lon, double lat){
    double r;

    lat = lat * M_PI / 180.0;
    lon = lon * M_PI / 180.0;

    p->x = sin(lon) * cos(lat);
    p->y = cos(lon) * cos(lat);
    p->z = sin(lat);

    r = pow(p->x, 2.0) + pow(p->y, 2.0) + pow(p->z, 2.0);
    r = sqrt(r);

    p->x /= r;
    p->y /= r;
    p->z /= r;
}